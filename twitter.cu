#include "hip/hip_runtime.h"
#include "util.c"

#define TREND_FETCH_TIME (5 * 60 * 1000) // should be 5 min
#define THREADS_PER_BLOCK 64


char* read_tweet(FILE * stream);
size_t read_trends(char ** trends, FILE * file);

//TODO: RENAME trend_maps, gpu_matrix, gpu_hashed_words


/* 
 * Populate a NUMTWEETS x word_count matrix with word counts per tweet
 *
 * @post gpu_matrix[i * word_count + j] == # of occurrences of word j in tweet i
 *
 * gpu_tweets - the compressed representation of tweets
 * gpu_hashed_words - the compressed representation of words
 * TODO: these words aren't actually hashed, are they?
 * gpu_matrix - the resulting matrix of word counts per tweet 
 * word_count - the number of distinct words in the current batch
 */
__global__ void compute_word_containment(int * gpu_tweets, 
                                         int * gpu_hashed_words,
                                         char * gpu_matrix,
                                         int word_count);

/* 
 * Populate a NUMTRENDS x word_count matrix with word counts per trend
 *
 * @post trend_maps[i * word_count + j] == number of occurrences of word j in
 *                                         tweets that contain trend i
 *
 * trend_maps - a NUMTRENDS x word_count matrix, denoting word counts per trend
 * gpu_tweets_in_trends - array of number of tweets containing each trend 
 * gpu_trends - the compressed representation of trends
 * word_count - the number of distinct words in the batch
 * gpu_matrix - the matrix of word counts per tweet 
 */
__global__ void get_trend_word_counts(int * trend_maps,
                                      int * gpu_tweets_in_trends,
                                      int * gpu_trends,
                                      int word_count,
                                      char * gpu_matrix);

/*
 * Populate the matrix of words correlated with each trend


TODO: maybe combine with get_trend_word_counts? the resulting matrix is very
similar, has same dimensions

 * trend_maps - word counts for each trend
 * gpu_tweets_in_trends - number of tweets in a trend
 * total_word_counts - word counts for all words in all tweets
 * correlated_words - output, what's correlated
 * word_count - the number of distinct words in the batch
 */
__global__ void get_correlated_words(int * trend_maps,
                                     int * gpu_tweets_in_trends,
                                     int * total_word_counts,
                                     char * correlated_words,
                                     int word_count);


// Error-checking wrapper for hipMalloc
void * CudaMalloc(size_t size, char * error_message) {
  void ** ptr = NULL;
  if(hipMalloc(ptr, sizeof(int) * NUMTWEETS * COMPRESSEDLEN)
     != hipSuccess) {
    fprintf(stderr, "Failed to allocate %s on GPU\n", error_message);
    exit(2);
  }
  return *ptr;
}

// Error-checking wrapper for hipMemcpy
void CudaMemcpy(void * destination, void * source, size_t size,
                enum hipMemcpyKind direction, char * error_message) {
  if(hipMemcpy(destination, source, size, direction) != hipSuccess) {
    fprintf(stderr, "Failed to copy %s to the GPU\n", error_message);
  }
}


// Main function
int main(int argc, char** argv) {
  // Timer for trend fetching. Should be every 5 minutes
  size_t start_time = time_ms() - TREND_FETCH_TIME - 1;

  // The trends
  char ** trends = (char **)malloc(sizeof(char *) * NUMTRENDS);
  // The tweets
  char tweets[NUMTWEETS][TWEETSIZE];

  // Array of compressed tweets and trends on host and device
  int compressed_tweets[NUMTWEETS][COMPRESSEDLEN];
  int compressed_trends[NUMTRENDS];
  int * gpu_tweets = (int*)CudaMalloc(sizeof(int) * NUMTWEETS * COMPRESSEDLEN,
                                      "the tweets");
  /*
  int * gpu_tweets;
  if(hipMalloc(&gpu_tweets, sizeof(int) * NUMTWEETS * COMPRESSEDLEN)
     != hipSuccess) {
    fprintf(stderr, "Failed to allocate the tweets on GPU\n");
    exit(2);
  }
  */
  int * gpu_trends;
  if(hipMalloc(&gpu_tweets, sizeof(int) * NUMTRENDS) != hipSuccess) {
    fprintf(stderr, "Failed to allocate the trends on GPU\n");
    exit(2);
  }
  
  // Topic containment matrix
  char * trend_matrix = (char *)calloc(sizeof(char), // size of a cell
                                       sizeof(char) * NUMTWEETS * NUMTRENDS);
  char * gpu_matrix;
  if(hipMalloc(&gpu_tweets, sizeof(int) * NUMTWEETS * NUMTRENDS)
     != hipSuccess) {
    fprintf(stderr, "Failed to allocate the matrix on GPU\n");
    exit(2);
  }
  // word count maps for every trend
  int * trend_maps; //TODO zero out on every iteration

  // Word arrays
  char words[NUMTWEETS*COMPRESSEDLEN][TWEETSIZE];
  int hashed_words[NUMTWEETS*COMPRESSEDLEN]; // TODO we don't need this?
  int total_word_counts[NUMTWEETS*COMPRESSEDLEN];
  int word_count = 0;
  
  // The pipe for the tweet stream
  int fd_tweets[2];
  int fd_trends[2];
  FILE * tweet_stream;

  char* tweet_args[] = {"cat", "tweets.json", NULL};
  char* trend_args[] = {"cat", "trends.json", NULL};

  // an error checking thing for forks
  int rc;
  
  int tweet_count = 0;

  int first_iteration = 1;
  size_t num_trends;
  
  // Open the tweet stream
  if (pipe (fd_tweets) < 0){
    perror("pipe error");
    exit(1);
  }
  rc = fork();
  if (rc < 0) { // error
    perror("Fork failed!");
    exit(1);
  } else if (rc == 0) { // child    
    pipe_stream(tweet_args, fd_tweets);
  } else {
    // parent: open the pipe as a file and keep it alive
    close(fd_tweets[1]);
    tweet_stream = fdopen(fd_tweets[0], "r");
  } 

  // Get the first tweet
  char* tweet = read_tweet(tweet_stream);
  
  // stop when out of tweets or trends or the user quits???
  while(tweet != NULL) {
    
    // TODO: Get stream of tweets and trends using forks and pipes
    if ((time_ms() - start_time) > TREND_FETCH_TIME) {
      start_time = time_ms();

      // open the pipe
      if (pipe (fd_trends) < 0){
        perror("pipe error");
        exit(1);
      }

      rc = fork();
      if (rc < 0) { // error
        perror("Fork failed!");
        exit(1);
      } else if (rc == 0) { // child
        pipe_stream(trend_args, fd_trends);
      } else { // parent
        close(fd_trends[1]);
        // read trends from stdout
        if (!first_iteration) {
          for (int i = 0; i < num_trends; i++) 
            free(trends[i]);
        } else { first_iteration = 0; }
        
        FILE * trend_stream = fdopen (fd_trends[0], "r");
        num_trends = read_trends(trends, trend_stream);
        fclose(trend_stream);
        if (num_trends < 1) {
          printf("Could not fetch trends\n");
          // TODO: cleanup
          exit(1);
        }
        // TESTING
        
        // Copy trends onto the GPU
        if(hipMemcpy(gpu_trends, compressed_trends, sizeof(int) * NUMTRENDS,
                      hipMemcpyHostToDevice) != hipSuccess) {
          fprintf(stderr, "Failed to copy trends to the GPU\n");
        }
      }
    }
    
    // save tweet by copying
    strncpy(tweets[tweet_count], tweet, TWEETSIZE);

    // TODO: Clean and compress the tweet
    clean_string(tweet);
    compress_str(tweet,
                 compressed_tweets[tweet_count],
                 words,
                 hashed_words,
                 total_word_counts,
                 &word_count);
    free(tweet);
    
    // TESTING
    printf("tweet #%d: %s\n", tweet_count, tweet);

    if (tweet_count >= NUMTWEETS - 1) {
      // Copy compressed tweets onto the GPU
      if(hipMemcpy(gpu_tweets, compressed_tweets,
                    sizeof(int) * NUMTWEETS * COMPRESSEDLEN,
                    hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy tweets to the GPU\n");
      }

      // Allocate trend maps
      if(hipMalloc(&trend_maps, sizeof(int) * word_count * NUMTRENDS)
         != hipSuccess) {
        fprintf(stderr, "Failed to allocate trend_maps on GPU\n");
        exit(2);
      }

      // Zero out trend_maps
      if(hipMemset(&trend_maps, 0, sizeof(int) * word_count * NUMTRENDS)
         != hipSuccess) {
        fprintf(stderr, "Failed to zero out trend_maps on GPU\n");
        exit(2);
      }

      // Figure out trends' compressed values (indices in the word array)
      for (int i = 0; i < num_trends; i++) {
        // go through words array to find each trend'd index
        for (int j = 0; j < word_count; j++) {
          if (strcmp(trends[i], words[j]) == 0) {
            compressed_trends[i] = j;
            break;
          }
        }
      }

      // TODO copy compressed_trends onto the GPU
      // TODO: copy word_count onto GPU
      // TODO: copy hashed_words onto GPU
      // TODO: copy words onto GPU
      // char words[NUMTWEETS*COMPRESSEDLEN][TWEETSIZE];
      // TODO: create gpu_tweets_in_trends on GPU & zero out

      // Zero out gpu_tweets
      
      // Make tweets x words matrix of counts. (similar to the thing below)  
      // TODO: Make an NxK topic containment bit matrix
      compute_word_containment<<<1, NUMTWEETS>>>(gpu_tweets, gpu_hashed_words,
                                                 gpu_matrix, word_count);

      // TODO: That syncing thing
      
      // To make trend_maps, add rows of the tweet X word matrix that correpond
      // to each trend
      // TODO: Get word counts for each tweet with a specific trend
      get_trend_word_counts<<<1, NUMTRENDS>>>(trend_maps,
                                              gpu_tweets_in_trends,
                                              gpu_trends,
                                              word_count,
                                              gpu_matrix);
      
      // TODO: Find word sets correlated with each topic and compute correlation
      //   coefficients
      get_correlated_words<<<1, NUMTRENDS>>>(trend_maps,
                                             gpu_tweets_in_trends,
                                             total_word_counts,
                                             correlated_words,
                                             word_count);

      
      // TODO: Create word clouds with external tools (go to 8 if it doesn’t
      //   work out)
      // TODO: If time allows: Implement weighing words by importance (tf-idf)
      // TODO: If time allows: Explore other uses of the same output data: graph
      //   building, clustering, or term evolution over time
      // TODO: If time allows: Rewrite compress_str to use insertion sort and binary search
      //   maybe with an auxiliary data structure

      // Free stuff: trend_maps
      // Zero out gpu_matrix
      
      
      word_count = 0;
      tweet_count = 0;
    } // for each NUMTWEETS tweets
    
    // Read the next tweet  
    tweet = read_tweet(tweet_stream);
    tweet_count++;
  }
  // Close the pipe and the file
  fclose(tweet_stream);
  close(fd_tweets[0]);

  // Free CUDA stuff
  
  //for (int i = 0; i < NUMTWEETS; i++)
  //  free(tweets[i]);
  //free(tweets);
  
  for (int i = 0; i < NUMTRENDS; i++)
    free(trends[i]);
  free(trends);
  
  return 0;
}

/* Get inputs from the feed */

// Returns the size of the trends array
// Make sure to allocate the trends array!
// char ** trends = (char **)malloc(sizeof(char *) * NUMTRENDS);
size_t read_trends(char ** trends, FILE * file) {

  // for using getline
  static char* line = NULL;
  static size_t line_maxlen = 0;
  
  // Loop until we read one valid json array or reach the end of the input
  while(getline(&line, &line_maxlen, file) > 0) {
    // Parse the JSON body
    json_error_t error;
    // The outer array, hypothetically
    json_t* root = json_loads(line, 0, &error);
  
    // Skip over lines with errors
    if(!root) continue;
  
    // Skip over lines that aren't JSON nonempty arrays
    if(!json_is_array(root) || json_array_size(root) < 1) {
      json_decref(root);
      continue;
    }

    // Get the first object in the array
    json_t * first_object = json_array_get(root, 0);

    // Get the json trends array
    json_t * json_trends_array = json_object_get(first_object, "trends");

    size_t arr_size;
     // Make sure 'trends' is a nonempty array
    if(!json_is_array(json_trends_array) ||
       (arr_size = json_array_size(json_trends_array)) < 1) {
      json_decref(json_trends_array);
      json_decref(first_object);
      json_decref(root);
      continue;
    }

    size_t i;
    // Read every trend into a regular C array
    for (i = 0; i < arr_size &&  i < NUMTRENDS; i++) {
      json_t * json_trend_obj = json_array_get(json_trends_array, i);
      
      // Get the name of the trend
      json_t* text = json_object_get(json_trend_obj, "name");
  
      // If there was no text, skip this t
      if(!json_is_string(text)) {
        json_decref(root);
        continue;
      }
  
      // Get the string out of the JSON text value
      const char* json_text = json_string_value(text);
  
      // Got a trend! Copy just the trend text to an allocated buffer
      trends[i] = (char*)malloc(sizeof(char) * (strlen(json_text) + 1));
      strcpy(trends[i], json_text);
    
      // Release this reference to the JSON object
      json_decref(text);
      json_decref(json_trend_obj);
    }

    // Release references to JSON objects
    json_decref(json_trends_array);
    json_decref(first_object);
    json_decref(root);

    //free(line);
    
    // Return the number of trends read
    return i;
  }
  
  // Ran out of input. Just return 0
  return 0;
} // read_trends


char* read_tweet(FILE * stream) {
  static char* line = NULL;
  static size_t line_maxlen = 0;
  ssize_t line_length;
  
  // Loop until we read one valid tweet or reach the end of the input
  while((line_length = getline(&line, &line_maxlen, stream)) > 0) {
    // Parse the JSON body
    json_error_t error;
    json_t* root = json_loads(line, 0, &error);
  
    // Skip over lines with errors
    if(!root) continue;
  
    // Skip over lines that aren't JSON objects
    if(!json_is_object(root)) {
      json_decref(root);
      continue;
    }
  
    // Get the text of the tweet
    json_t* text = json_object_get(root, "text");
  
    // If there was no text, skip this tweet
    if(!json_is_string(text)) {
      json_decref(root);
      continue;
    }
  
    // Get the string out of the JSON text value
    const char* json_text = json_string_value(text);
  
    // Got a tweet! Copy just the tweet text to an allocated buffer
    char* tweet_text = (char*)malloc(sizeof(char) * (line_length + 1));
    strcpy(tweet_text, json_text);
    
    // Release this reference to the JSON object
    json_decref(root);

    //free(line);
    
    // Return the result
    return tweet_text;
  }
  
  // Ran out of input. Just return NULL
  return NULL;
}



/* CUDA functions */

__device__ void get_intersect(int *tweet1,int *tweet2, int *intersect){
  intersect = (int*) malloc(sizeof(int) * (COMPRESSEDLEN));
  int i = 0; // tweet1
  int j = 0; // tweet2
  int index_intersect;
  int k = 0; // For the intersection
  while (tweet1[i] != 0) { 
    while (tweet2[j] != 0) {
      if (tweet2[j] == tweet1[i]) {
        index_intersect = 0;
        while (index_intersect < k) {
          // check that we have no repeats in our intersection
          if (tweet1[i] == intersect[index_intersect]) {
            break;
          }
        }
        // if it didn't find the tweet in the intersection, add it
        if (index_intersect == k) {
          intersect[k++] = tweet1[i];
        }
      }
      j++;
    }
    i++;
    j = 0;
  }
}

// N tweets x K trends
// gpu_matrix must be zeroed out
__global__ void compute_word_containment(int * gpu_tweets,
                                         int * gpu_hashed_words,
                                         char * gpu_matrix,
                                         int word_count) {
  int index =  threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  if (index < NUMTWEETS) {
    int * tweet = gpu_tweets[COMPRESSEDLEN * index];
    for (int i = 0; i < COMPRESSEDLEN && tweet[i] != 0; i++) {
      for (int j = 0; j < word_count; j++) {
        if (tweet[i] == gpu_hashed_words[j]) // should be just  == j
          gpu_matrix[word_count * index + j]++;
        /* //Can't we just say this, since tweet[i] *is* the index of the word
           gpu_matrix[word_count * index + tweet[i]]++;
         */
      }
    }
  }
}

__global__ void get_trend_word_counts(int * trend_maps,
                                      int * gpu_tweets_in_trends,
                                      int * gpu_trends,
                                      int word_count,
                                      char * gpu_matrix) {
  int trend_index =  threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  if (trend_index < NUMTRENDS) {
    int trend_map_index = trend_index * word_count;
    int trend_word_index = gpu_trends[trend_index];
    for (int i = 0; i < NUMTWEETS; i++) { // for every tweet
      if (gpu_matrix[i * word_count + trend_word_index] > 0) {
        // if the trend is present in the tweet
        for (int j = 0; j < word_count; j++) {
          // get the word counts for all the words in the tweet
          trend_maps[trend_map_index + j] +=
            gpu_matrix[i * word_count + j];
        }
        gpu_tweets_in_trend[trend_index]++;
      }
    }
  }
}

__global__ void get_correlated_words(int * trend_maps,
                                     int * gpu_tweets_in_trends,
                                     int * total_word_counts,
                                     char * correlated_words,
                                     int word_count) {
  int trend_index =  threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  if (trend_index < NUMTRENDS) {
    for (int i = 0; i < word_count; i++) {
      if (trend_maps[word_count * trend_index + i] /
          (double) gpu_tweets_in_trends[trend_index] >
          CONSTANT * total_word_counts[i] / (double) word_count) {
        correlated_words[word_count * trend_index + i] = 1;
      } else {
        correlated_words[word_count * trend_index + i] = 0;
      }
    }
  }
}
